#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"
// ===== Standard Attention Forward Pass Benchmark =====
void run_mha_fwd(nvbench::state& state) {
    hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
    state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));
  
    int64_t num_seqs = state.get_int64("num_seqs");
    int64_t seq_len = state.get_int64("seq_len");
    int64_t num_heads = state.get_int64("num_heads");
    int64_t num_kv_heads = state.get_int64("num_kv_heads");
    int64_t head_size = state.get_int64("head_size");
    
    bool is_causal = state.get_int64_or_default("causal", 1) != 0;
    float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
    float softcap = state.get_float64_or_default("softcap", 0.0f);
    float dropout_p = state.get_float64_or_default("dropout", 0.0f);
    int64_t window_size_left = state.get_int64_or_default("window_left", -1);
    int64_t window_size_right = state.get_int64_or_default("window_right", -1);
    bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;
  
    auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
    
    try {
      // Create tensors for q, k, v with appropriate shapes
      auto q = torch::rand({num_seqs, seq_len, num_heads, head_size}, tensor_options);
      auto k = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
      auto v = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
      
      c10::optional<at::Tensor> out_ = std::nullopt;
      c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
      c10::optional<at::Generator> gen_ = std::nullopt;
      
      int sizeof_half = 2;
      int64_t o_write = num_seqs * seq_len * num_heads * head_size * sizeof_half;
      int64_t q_read = num_seqs * seq_len * num_heads * head_size * sizeof_half;
      int64_t k_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
      int64_t v_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
      
      // Add memory reads and writes
      state.add_global_memory_reads<char>(q_read + k_read + v_read, "Memory Reads");
      state.add_global_memory_writes<char>(o_write, "Memory Writes");
      
      // Memory usage estimation
      int64_t est_memory_usage = 
          (num_seqs * seq_len * num_heads * head_size * sizeof_half) +      // Query
          (num_seqs * seq_len * num_kv_heads * head_size * sizeof_half) +   // Key
          (num_seqs * seq_len * num_kv_heads * head_size * sizeof_half) +   // Value
          (num_seqs * seq_len * num_heads * head_size * sizeof_half);       // Output
      
      // Create proper summary
      auto& mem_summary = state.add_summary("memory_usage");
      mem_summary.set_string("name", "Memory Usage");
      mem_summary.set_string("description", "Estimated memory used in MiB");
      mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
      
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
        return;
      }
  
      // Total tokens for throughput calculation
      int64_t total_tokens = num_seqs * seq_len;
      state.add_element_count(total_tokens, "Tokens");
  
      state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
        timer.start();
        auto result = mha_fwd(
          q,
          k,
          v,
          out_,
          alibi_slopes_,
          dropout_p,
          softmax_scale,
          is_causal,
          window_size_left,
          window_size_right,
          softcap,
          return_softmax,
          gen_
        );
        auto status = hipGetLastError();
        if (status != hipSuccess) {
          state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
        }
        timer.stop();
      });
      
    } catch (const c10::Error& e) {
      state.skip("OOM: " + std::string(e.what()));
      return;
    }
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)
// Standard Attention Benchmark
NVBENCH_BENCH(run_mha_fwd)
    .set_name("run_mha_fwd")
    .add_int64_axis("num_seqs", {1, 4, 16, 32, 64})
    .add_int64_axis("seq_len", {512, 1024, 2048, 4096, 8192, 16384, 32768})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})  // Added 12, 40, 128
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 32, 128})  // Added 10, 12, 128
    .add_int64_axis("head_size", {56, 64, 80, 128});  // Added 56
