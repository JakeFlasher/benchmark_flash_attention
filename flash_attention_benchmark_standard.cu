#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include <sstream>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

void run_mha_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  float dropout_p = state.get_float64_or_default("dropout", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;

  // Report operation being benchmarked
  if (state.get_int64_or_default("verbose", 0)) {
    auto& op_summ = state.add_summary("operation_details");
    op_summ.set_string("name", "Standard Attention");
    
    std::stringstream desc_ss;
    desc_ss << "Q[" << num_seqs << "×" << seq_len << "×" << num_heads << "×" << head_size 
            << "] × K[" << num_seqs << "×" << seq_len << "×" << num_kv_heads << "×" << head_size 
            << "] → Out[" << num_seqs << "×" << seq_len << "×" << num_heads << "×" << head_size << "]";
    op_summ.set_string("description", desc_ss.str());
    
    if (is_causal) op_summ.set_string("mask", "causal");
    
    if (window_size_left > 0 || window_size_right > 0) {
      std::stringstream window_ss;
      window_ss << window_size_left << ":" << window_size_right;
      op_summ.set_string("window", window_ss.str());
    }
    
    op_summ.set_float64("gqa_ratio", static_cast<double>(num_heads) / num_kv_heads);
  }

  auto tensor_options = torch::TensorOptions()
                          .device(torch::kCUDA)
                          .dtype(torch::kFloat16);
  
  try {
    // Create tensors for q, k, v with appropriate shapes
    auto q = torch::rand({num_seqs, seq_len, num_heads, head_size}, tensor_options);
    auto k = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
    auto v = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
    
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    c10::optional<at::Generator> gen_ = std::nullopt;
    
    // Properly report memory operations and throughput
    int sizeof_datatype = 2; // Half precision = 2 bytes
    int64_t tokens_per_sec = num_seqs * seq_len;
    int64_t q_size = num_seqs * seq_len * num_heads * head_size * sizeof_datatype;
    int64_t k_size = num_seqs * seq_len * num_kv_heads * head_size * sizeof_datatype;
    int64_t v_size = num_seqs * seq_len * num_kv_heads * head_size * sizeof_datatype;
    int64_t out_size = num_seqs * seq_len * num_heads * head_size * sizeof_datatype;
    
    // Memory operations for attention (read Q,K,V, write output)
    state.add_global_memory_reads<char>(q_size, "Q Tensor");
    state.add_global_memory_reads<char>(k_size, "K Tensor");
    state.add_global_memory_reads<char>(v_size, "V Tensor");
    state.add_global_memory_writes<char>(out_size, "Output");
    
    // Token throughput
    state.add_element_count(tokens_per_sec, "Tokens");
    
    // Total floating point operations estimate 
    // (simplified - actual count depends on attention implementation)
    int64_t flops = 2 * num_seqs * seq_len * seq_len * num_heads * head_size;
    auto& flops_summ = state.add_summary("flops");
    flops_summ.set_string("name", "Est. FLOPS");
    flops_summ.set_string("description", "Estimated floating point operations");
    flops_summ.set_int64("value", flops);
    
    // Memory footprint
    int64_t est_memory_usage = q_size + k_size + v_size + out_size;
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      std::string error_msg = "CUDA error: ";
      error_msg += hipGetErrorString(status);
      state.skip(error_msg);
      return;
    }

    // Execute with proper timing
    state.exec([&](nvbench::launch& launch) {
      // Use proper stream access
      auto result = mha_fwd(
        q,
        k,
        v,
        out_,
        alibi_slopes_,
        dropout_p,
        softmax_scale,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        return_softmax,
        gen_
      );
      
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        std::string error_msg = "CUDA error: ";
        error_msg += hipGetErrorString(status);
        state.skip(error_msg);
      }
    });
    
  } catch (const c10::Error& e) {
    std::string error_msg = "OOM: ";
    error_msg += e.what();
    state.skip(error_msg);
    return;
  }
}

// Register the benchmark
NVBENCH_BENCH(run_mha_fwd)
    .set_name("run_mha_fwd")
    .add_int64_axis("num_seqs", {1, 4, 16, 32, 64})
    .add_int64_axis("seq_len", {512, 1024, 2048, 4096, 8192})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 32, 128})
    .add_int64_axis("head_size", {56, 64, 80, 128})
    .add_int64_axis("window_left", {-1, 128, 1024, 4096})
    .add_int64_axis("window_right", {-1, 128, 1024, 4096})
    .add_int64_axis("causal", {0, 1});
