#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"
// ===== Standard Attention Forward Pass Benchmark =====
void run_mha_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  // Get parameters
  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  
  // Get optional parameters with defaults
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  float dropout_p = state.get_float64_or_default("dropout", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;
  
  // Select data type based on parameter
  auto dtype = state.get_string_or_default("dtype", "float16") == "bfloat16" 
               ? torch::kBFloat16 : torch::kFloat16;
  
  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(dtype);
  
  try {
    // Create tensors for q, k, v with appropriate shapes
    auto q = torch::rand({num_seqs, seq_len, num_heads, head_size}, tensor_options);
    auto k = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
    auto v = torch::rand({num_seqs, seq_len, num_kv_heads, head_size}, tensor_options);
    
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    c10::optional<at::Generator> gen_ = std::nullopt;
    
    // Calculate sizes for memory reporting
    int64_t o_write_elements = num_seqs * seq_len * num_heads * head_size;
    int64_t q_read_elements = num_seqs * seq_len * num_heads * head_size;
    int64_t k_read_elements = num_seqs * seq_len * num_kv_heads * head_size;
    int64_t v_read_elements = num_seqs * seq_len * num_kv_heads * head_size;
    
    // Add memory reads and writes using proper NVBench methods
    if (dtype == torch::kFloat16) {
      state.add_global_memory_reads<at::Half>(q_read_elements + k_read_elements + v_read_elements);
      state.add_global_memory_writes<at::Half>(o_write_elements);
    } else {
      state.add_global_memory_reads<at::BFloat16>(q_read_elements + k_read_elements + v_read_elements);
      state.add_global_memory_writes<at::BFloat16>(o_write_elements);
    }
    
    // Add custom memory usage summary
    int64_t element_size = 2; // Both FP16 and BF16 are 2 bytes
    int64_t est_memory_usage = 
        (q_read_elements + k_read_elements + v_read_elements + o_write_elements) * element_size;
    
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    // Check for CUDA errors
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      return;
    }

    // Report tokens for throughput calculation
    int64_t total_tokens = num_seqs * seq_len;
    state.add_element_count(total_tokens, "Tokens");

    // Execute and time the kernel
    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto result = mha_fwd(
        q,
        k,
        v,
        out_,
        alibi_slopes_,
        dropout_p,
        softmax_scale,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        return_softmax,
        gen_
      );
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      }
      timer.stop();
    });
    
  } catch (const c10::Error& e) {
    state.skip("OOM: " + std::string(e.what()));
    return;
  }
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x) 

NVBENCH_BENCH(run_mha_fwd)
    .set_name("run_mha_fwd")
    .add_int64_axis("num_seqs", {1, 4, 16, 32, 64})
    .add_int64_axis("seq_len", {512, 1024, 2048, 4096, 8192, 16384, 32768})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 32, 128})
    .add_int64_axis("head_size", {56, 64, 80, 96, 128, 160})
    .add_string_axis("dtype", {"float16", "bfloat16"});

