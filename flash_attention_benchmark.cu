#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>

#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

void run_mha_varlen_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  constexpr int64_t max_tokens_in_benchmark = 1048576;  // 2^20

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  int64_t page_size = state.get_int64("page_size");

  int64_t max_tokens_per_seq = max_tokens_in_benchmark / num_seqs;
  int64_t max_pages_per_seq = max_tokens_per_seq / page_size;

  if (seq_len > max_tokens_per_seq) {
    state.skip("seq_len too long to fit into the benchmark preset");
    return;
  }
  int max_context_len = seq_len;

  int64_t num_pages = ceil_div(max_tokens_in_benchmark, page_size);

  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
  // auto o = torch::empty({num_seqs, 1, num_heads, head_size}, tensor_options);
  auto q = torch::empty({num_seqs, 1, num_heads, head_size}, tensor_options);
  auto k_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
  auto v_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
  auto seq_lens = torch::full({num_seqs}, seq_len, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));

  // fill page_table with unique mapping
  auto page_table_1d = torch::arange(0, num_seqs * max_pages_per_seq, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));
  auto page_table = torch::reshape(page_table_1d, {num_seqs, max_pages_per_seq});

  float scale = 1.0f / sqrt(num_heads);

  int sizeof_half = 2;
  int64_t o_write = num_seqs * 1 * num_heads * head_size * sizeof_half;
  int64_t q_read = num_seqs * 1 * num_heads * head_size * sizeof_half;
  int64_t k_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
  int64_t v_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
  state.add_global_memory_reads<char>(q_read + k_read + v_read, "Read");
  state.add_global_memory_writes<char>(o_write, "Write");

  auto status = hipGetLastError();
  if (status != hipSuccess) {
    state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
  }

  c10::optional<const at::Tensor> null_const_tensor = std::nullopt;
  c10::optional<const at::Tensor> opt_seq_lens = seq_lens;
  c10::optional<at::Tensor> null_mutable_tensor = std::nullopt;
  c10::optional<at::Tensor> opt_page_table = page_table;
  // c10::optional<at::Tensor> opt_o = o;

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
    timer.start();
    // https://github.com/Dao-AILab/flash-attention/blob/c92ca63268836a6ea519e9892eda4895fc31811d/flash_attn/flash_attn_interface.py#L1207-L1244
    auto out = mha_fwd_kvcache(
      q,
      k_cache,
      v_cache,
      null_const_tensor,
      null_const_tensor,
      opt_seq_lens,
      null_const_tensor,
      null_const_tensor,
      null_const_tensor,
      null_const_tensor,
      opt_page_table,
      null_mutable_tensor,
      null_mutable_tensor,
      scale,
      true,
      -1,
      -1,
      0.0f,
      true,
      0
    );
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
    }
    timer.stop();
  });
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)

NVBENCH_BENCH(run_mha_varlen_fwd)
    .set_name(("run_mha_varlen_fwd"))
    .add_int64_axis("num_seqs", {1, 4, 16, 64})
    .add_int64_axis("seq_len", {32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536})
    .add_int64_axis("num_heads", {32, 64})
    .add_int64_axis("num_kv_heads", {32, 8, 4, 2})
    .add_int64_axis("head_size", {96, 128})
    .add_int64_axis("page_size", {256});
