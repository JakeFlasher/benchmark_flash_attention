#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include <sstream>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

void run_mha_kvcache(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  // Configuration parameters
  int64_t max_tokens_in_benchmark = state.get_int64_or_default("max_tokens", 1048576);  // 2^20

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  int64_t page_size = state.get_int64("page_size");
  
  // Other parameters
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool is_rotary_interleaved = state.get_int64_or_default("rotary_interleaved", 1) != 0;
  int num_splits = state.get_int64_or_default("num_splits", 0);

  // Add operation description
  if (state.get_int64_or_default("verbose", 0)) {
    auto& op_summ = state.add_summary("operation_details");
    op_summ.set_string("name", "KV Cache Attention");
    
    std::stringstream desc_ss;
    desc_ss << "Q[" << num_seqs << "×1×" << num_heads << "×" << head_size 
            << "] with KV cache[" << seq_len << "×" << page_size << "]";
    op_summ.set_string("description", desc_ss.str());
    
    if (is_causal) op_summ.set_string("mask", "causal");
    
    if (window_size_left > 0 || window_size_right > 0) {
      std::stringstream window_ss;
      window_ss << window_size_left << ":" << window_size_right;
      op_summ.set_string("window", window_ss.str());
    }
    
    op_summ.set_float64("gqa_ratio", static_cast<double>(num_heads) / num_kv_heads);
    op_summ.set_int64("page_size", page_size);
  }

  // Cache allocation limits
  int64_t max_tokens_per_seq = max_tokens_in_benchmark / num_seqs;
  int64_t max_pages_per_seq = max_tokens_per_seq / page_size;

  if (seq_len > max_tokens_per_seq) {
    state.skip("seq_len too long to fit into the benchmark preset");
    return;
  }
  
  int64_t num_pages = ceil_div(max_tokens_in_benchmark, page_size);
  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
  
  try {
    // Create tensors for query and cache
    auto q = torch::empty({num_seqs, 1, num_heads, head_size}, tensor_options);
    auto k_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto v_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto seq_lens = torch::full({num_seqs}, seq_len, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));

    // Fill page_table with unique mapping
    auto page_table_1d = torch::arange(0, num_seqs * max_pages_per_seq, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));
    auto page_table = torch::reshape(page_table_1d, {num_seqs, max_pages_per_seq});

    // Memory size tracking
    int sizeof_half = 2;
    int64_t q_size = num_seqs * 1 * num_heads * head_size * sizeof_half;
    int64_t k_cache_size = num_pages * page_size * num_kv_heads * head_size * sizeof_half;
    int64_t v_cache_size = num_pages * page_size * num_kv_heads * head_size * sizeof_half;
    int64_t out_size = num_seqs * 1 * num_heads * head_size * sizeof_half;
    int64_t page_table_size = num_seqs * max_pages_per_seq * 4; // Int32 indices
    
    // Memory operations tracking
    state.add_global_memory_reads<char>(q_size, "Query");
    state.add_global_memory_reads<char>(k_cache_size, "K Cache");
    state.add_global_memory_reads<char>(v_cache_size, "V Cache");
    state.add_global_memory_writes<char>(out_size, "Output");
    
    // Tokens per second
    state.add_element_count(num_seqs, "Tokens/Step");
    
    // Context size in tokens
    auto& ctx_summ = state.add_summary("context_length");
    ctx_summ.set_string("name", "Context Len");
    ctx_summ.set_string("description", "Context length in tokens");
    ctx_summ.set_int64("value", seq_len);
    
    // Memory usage estimation
    int64_t est_memory_usage = q_size + k_cache_size + v_cache_size + out_size + page_table_size;
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    // KV cache specific metrics
    auto& cache_summary = state.add_summary("kv_cache");
    cache_summary.set_string("name", "KV Cache");
    cache_summary.set_string("description", "KV Cache configuration");
    cache_summary.set_int64("page_size", page_size);
    cache_summary.set_int64("num_pages", num_pages);
    cache_summary.set_int64("pages_per_seq", max_pages_per_seq);
    cache_summary.set_float64("size_mib", (k_cache_size + v_cache_size) / (1024.0 * 1024.0));
    cache_summary.set_int64("value", num_pages * page_size); // Total KV cache capacity in tokens
    // FLOPS estimation
int64_t flops = 2 * num_seqs * seq_len * num_heads * head_size;
auto& flops_summ = state.add_summary("flops");
flops_summ.set_string("name", "Est. FLOPS");
flops_summ.set_string("description", "Estimated floating point operations");
flops_summ.set_int64("value", flops);    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      std::string error_msg = "CUDA error: ";
      error_msg += hipGetErrorString(status);
      state.skip(error_msg);
      return;
    }

    // Optional tensors
    c10::optional<const at::Tensor> null_const_tensor = std::nullopt;
    c10::optional<const at::Tensor> opt_seq_lens = seq_lens;
    c10::optional<at::Tensor> null_mutable_tensor = std::nullopt;
    c10::optional<at::Tensor> opt_page_table = page_table;

    // Execute benchmark with simple launch
    state.exec([&](nvbench::launch& launch) {
      auto out = mha_fwd_kvcache(
        q,
        k_cache,
        v_cache,
        null_const_tensor,
        null_const_tensor,
        opt_seq_lens,
        null_const_tensor,
        null_const_tensor,
        null_const_tensor,
        null_const_tensor,
        opt_page_table,
        null_mutable_tensor,
        null_mutable_tensor,
        softmax_scale,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        is_rotary_interleaved,
        num_splits
      );
      
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        std::string error_msg = "CUDA error: ";
        error_msg += hipGetErrorString(status);
        state.skip(error_msg);
      }
    });
    
  } catch (const c10::Error& e) {
    std::string error_msg = "OOM: ";
    error_msg += e.what();
    state.skip(error_msg);
    return;
  }
}

// Register benchmark with expanded configuration parameters
NVBENCH_BENCH(run_mha_kvcache)
    .set_name("run_mha_kvcache")
    .add_int64_axis("num_seqs", {1, 2, 4, 8, 16, 32, 64})
    .add_int64_axis("seq_len", {32, 128, 512, 1024, 2048, 4096, 8192, 16384, 32768})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})
    .add_int64_axis("num_kv_heads", {1, 4, 8, 10, 12, 16, 32, 64, 128})
    .add_int64_axis("head_size", {56, 64, 80, 96, 128, 160})
    .add_int64_axis("page_size", {128, 256, 512, 1024, 2048})
    .add_int64_axis("window_left", {-1, 128, 1024, 4096})
    .add_int64_axis("window_right", {-1, 128, 1024, 4096})
    .add_int64_axis("causal", {0, 1});
