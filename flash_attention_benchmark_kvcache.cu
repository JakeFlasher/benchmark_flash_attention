#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}
// Your existing KV cache benchmark function
void run_mha_varlen_fwd_kvcache(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  int64_t max_tokens_in_benchmark = state.get_int64_or_default("max_tokens", 1048576);  // 2^20

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  int64_t page_size = state.get_int64("page_size");
  
  // Get optional parameters with defaults
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool is_rotary_interleaved = state.get_int64_or_default("rotary_interleaved", 1) != 0;
  int num_splits = state.get_int64_or_default("num_splits", 0);
  
  // Select data type based on parameter
  auto dtype = state.get_string_or_default("dtype", "float16") == "bfloat16" 
               ? torch::kBFloat16 : torch::kFloat16;

  int64_t max_tokens_per_seq = max_tokens_in_benchmark / num_seqs;
  int64_t max_pages_per_seq = max_tokens_per_seq / page_size;

  if (seq_len > max_tokens_per_seq) {
    state.skip("seq_len too long to fit into the benchmark preset");
    return;
  }

  int64_t num_pages = ceil_div(max_tokens_in_benchmark, page_size);
  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(dtype);
  
  try {
    auto q = torch::empty({num_seqs, 1, num_heads, head_size}, tensor_options);
    auto k_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto v_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto seq_lens = torch::full({num_seqs}, seq_len, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));

    // Generate page table - unique mapping
    auto page_table_1d = torch::arange(0, num_seqs * max_pages_per_seq, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));
    auto page_table = torch::reshape(page_table_1d, {num_seqs, max_pages_per_seq});

    // Optional: Add rotary embeddings if testing those
    c10::optional<at::Tensor> rotary_cos = std::nullopt;
    c10::optional<at::Tensor> rotary_sin = std::nullopt;
    
    // Conditionally create rotary embeddings based on parameter
    if (state.get_int64_or_default("use_rope", 0) != 0) {
      int64_t rot_dim = state.get_int64_or_default("rot_dim", head_size);
      if (rot_dim > head_size) rot_dim = head_size;
      if (rot_dim % 2 != 0) rot_dim -= 1;  // Must be even
      
      auto rope_theta = state.get_float64_or_default("rope_theta", 10000.0);
      auto max_position = state.get_int64_or_default("max_position", seq_len * 2);
      
      // Create proper RoPE tensors
      auto positions = torch::arange(max_position, torch::TensorOptions().device(torch::kCUDA));
      auto dim_indices = torch::arange(0, rot_dim/2, torch::TensorOptions().device(torch::kCUDA));
      auto inv_freq = 1.0 / torch::pow(rope_theta, (2.0 * dim_indices) / rot_dim);
      
      auto sincos_input = torch::einsum("i,j->ij", {positions, inv_freq});
      rotary_cos = torch::cos(sincos_input).to(dtype);
      rotary_sin = torch::sin(sincos_input).to(dtype);
    }

    // Calculate memory stats for reporting
    int element_size = 2;  // Both FP16 and BF16 are 2 bytes
    int64_t o_write = num_seqs * 1 * num_heads * head_size * element_size;
    int64_t q_read = num_seqs * 1 * num_heads * head_size * element_size;
    int64_t k_read = num_seqs * seq_len * num_kv_heads * head_size * element_size;
    int64_t v_read = num_seqs * seq_len * num_kv_heads * head_size * element_size;
    
    // Add memory reads and writes for throughput calculations
    if (dtype == torch::kFloat16) {
      state.add_global_memory_reads<at::Half>(q_read/element_size + k_read/element_size + v_read/element_size);
      state.add_global_memory_writes<at::Half>(o_write/element_size);
    } else {
      state.add_global_memory_reads<at::BFloat16>(q_read/element_size + k_read/element_size + v_read/element_size);
      state.add_global_memory_writes<at::BFloat16>(o_write/element_size);
    }
    
    // Memory usage estimation for custom summary
    int64_t est_memory_usage = 
        (num_seqs * 1 * num_heads * head_size * element_size) +
        (2 * num_pages * page_size * num_kv_heads * head_size * element_size) +
        (num_seqs * max_pages_per_seq * 4);
    
    if (rotary_cos.has_value()) {
      est_memory_usage += rotary_cos.value().numel() * element_size;
      est_memory_usage += rotary_sin.value().numel() * element_size;
    }
    
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used by the model in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      return;
    }

    c10::optional<const at::Tensor> null_const_tensor = std::nullopt;
    c10::optional<const at::Tensor> opt_seq_lens = seq_lens;
    c10::optional<at::Tensor> null_mutable_tensor = std::nullopt;
    c10::optional<at::Tensor> opt_page_table = page_table;

    // Report tokens for throughput calculations
    int64_t total_tokens = num_seqs * seq_len;
    state.add_element_count(total_tokens, "Tokens");

    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto out = mha_fwd_kvcache(
        q,
        k_cache,
        v_cache,
        null_const_tensor,
        null_const_tensor,
        opt_seq_lens,
        rotary_cos,
        rotary_sin,
        null_const_tensor,
        null_const_tensor,
        opt_page_table,
        null_mutable_tensor,
        null_mutable_tensor,
        softmax_scale,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        is_rotary_interleaved,
        num_splits
      );
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      }
      timer.stop();
    });
    
  } catch (const c10::Error& e) {
    state.skip("OOM: " + std::string(e.what()));
    return;
  }
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)
// KV Cache Benchmark
NVBENCH_BENCH(run_mha_varlen_fwd_kvcache)
    .set_name("run_mha_varlen_fwd_kvcache")
    .add_int64_axis("num_seqs", {1, 2, 4, 8, 16, 32, 64})
    .add_int64_axis("seq_len", {512, 1024, 2048, 4096, 8192, 16384, 32768})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 32, 128})
    .add_int64_axis("head_size", {56, 64, 80, 96, 128, 160})
    .add_int64_axis("page_size", {128, 256, 512, 1024, 2048})
    .add_string_axis("dtype", {"float16", "bfloat16"})
    .add_int64_axis("use_rope", {0, 1});  // Add RoPE testing