#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}
// Your existing KV cache benchmark function
void run_mha_varlen_fwd_kvcache(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  // Replace fixed token limit with a parameter
  int64_t max_tokens_in_benchmark = state.get_int64_or_default("max_tokens", 1048576);  // 2^20

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  int64_t page_size = state.get_int64("page_size");
  
  // Using int64 for boolean flags instead
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool is_rotary_interleaved = state.get_int64_or_default("rotary_interleaved", 1) != 0;
  int num_splits = state.get_int64_or_default("num_splits", 0);

  int64_t max_tokens_per_seq = max_tokens_in_benchmark / num_seqs;
  int64_t max_pages_per_seq = max_tokens_per_seq / page_size;

  if (seq_len > max_tokens_per_seq) {
    state.skip("seq_len too long to fit into the benchmark preset");
    return;
  }
  int max_context_len = seq_len;

  int64_t num_pages = ceil_div(max_tokens_in_benchmark, page_size);

  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
  
  // Improved error handling to handle GPU OOM gracefully
  try {
    auto q = torch::empty({num_seqs, 1, num_heads, head_size}, tensor_options);
    auto k_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto v_cache = torch::empty({num_pages, page_size, num_kv_heads, head_size}, tensor_options);
    auto seq_lens = torch::full({num_seqs}, seq_len, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));

    // fill page_table with unique mapping
    auto page_table_1d = torch::arange(0, num_seqs * max_pages_per_seq, torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32));
    auto page_table = torch::reshape(page_table_1d, {num_seqs, max_pages_per_seq});

    int sizeof_half = 2;
    int64_t o_write = num_seqs * 1 * num_heads * head_size * sizeof_half;
    int64_t q_read = num_seqs * 1 * num_heads * head_size * sizeof_half;
    int64_t k_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
    int64_t v_read = num_seqs * seq_len * num_kv_heads * head_size * sizeof_half;
    
    // Add memory reads and writes
    state.add_global_memory_reads<char>(q_read + k_read + v_read, "Memory Reads");
    state.add_global_memory_writes<char>(o_write, "Memory Writes");
    
    // Memory usage estimation - with CORRECT summary setup
    int64_t est_memory_usage = 
        // Query memory
        (num_seqs * 1 * num_heads * head_size * sizeof_half) +
        // KV cache memory 
        (2 * num_pages * page_size * num_kv_heads * head_size * sizeof_half) +
        // Page table memory
        (num_seqs * max_pages_per_seq * 4);
    
    // Create a proper summary with all required fields
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");  // Required field
    mem_summary.set_string("description", "Estimated memory used by the model in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      return;
    }

    c10::optional<const at::Tensor> null_const_tensor = std::nullopt;
    c10::optional<const at::Tensor> opt_seq_lens = seq_lens;
    c10::optional<at::Tensor> null_mutable_tensor = std::nullopt;
    c10::optional<at::Tensor> opt_page_table = page_table;

    // Total tokens for throughput calculation
    int64_t total_tokens = num_seqs * seq_len;
    state.add_element_count(total_tokens, "Tokens");

    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto out = mha_fwd_kvcache(
        q,
        k_cache,
        v_cache,
        null_const_tensor,
        null_const_tensor,
        opt_seq_lens,
        null_const_tensor,
        null_const_tensor,
        null_const_tensor,
        null_const_tensor,
        opt_page_table,
        null_mutable_tensor,
        null_mutable_tensor,
        softmax_scale,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        is_rotary_interleaved,\
        num_splits
      );
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      }
      timer.stop();
    });
    
  } catch (const c10::Error& e) {
    state.skip("OOM: " + std::string(e.what()));
    return;
  }
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)
// KV Cache Benchmark
NVBENCH_BENCH(run_mha_varlen_fwd_kvcache)
    .set_name(("run_mha_varlen_fwd_kvcache"))
    .add_int64_axis("num_seqs", {1, 2, 4, 8, 16, 32, 64})
    .add_int64_axis("seq_len", {32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768})  // Added 32768
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 80, 128})  // Added 12, 40, 128
    .add_int64_axis("num_kv_heads", {1, 2, 4, 8, 10, 12, 16, 32, 64, 128})  // Added 10, 12, 128
    .add_int64_axis("head_size", {56, 64, 80, 96, 128, 160})  // Added 56
    .add_int64_axis("page_size", {128, 256, 512, 1024, 2048});  // Added 2048 for long context
