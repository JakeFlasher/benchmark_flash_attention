#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"
// ===== Variable Length Attention Forward Pass Benchmark =====
void run_mha_varlen_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  float dropout_p = state.get_float64_or_default("dropout", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;
  bool zero_tensors = state.get_int64_or_default("zero_tensors", 0) != 0;

  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
  auto int_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32);
  
  try {
    // For variable length sequences, we pack them into a single tensor
    int total_tokens = num_seqs * seq_len;
    
    // Create packed tensors
    auto q = torch::rand({total_tokens, num_heads, head_size}, tensor_options);
    auto k = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    auto v = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    
    // Create cumulative sequence lengths tensor
    auto cu_seqlens = torch::zeros({num_seqs + 1}, int_options);
    for (int i = 1; i <= num_seqs; i++) {
      cu_seqlens[i] = i * seq_len;
    }
    
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> seqused_k = std::nullopt;
    c10::optional<const at::Tensor> leftpad_k_ = std::nullopt;
    c10::optional<at::Tensor> block_table_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    c10::optional<at::Generator> gen_ = std::nullopt;
    
    int sizeof_half = 2;
    int64_t o_write = total_tokens * num_heads * head_size * sizeof_half;
    int64_t q_read = total_tokens * num_heads * head_size * sizeof_half;
    int64_t k_read = total_tokens * num_kv_heads * head_size * sizeof_half;
    int64_t v_read = total_tokens * num_kv_heads * head_size * sizeof_half;
    
    // Add memory reads and writes
    state.add_global_memory_reads<char>(q_read + k_read + v_read, "Memory Reads");
    state.add_global_memory_writes<char>(o_write, "Memory Writes");
    
    // Memory usage estimation
    int64_t est_memory_usage = 
        (total_tokens * num_heads * head_size * sizeof_half) +      // Query
        (total_tokens * num_kv_heads * head_size * sizeof_half) +   // Key
        (total_tokens * num_kv_heads * head_size * sizeof_half) +   // Value
        (total_tokens * num_heads * head_size * sizeof_half) +      // Output
        ((num_seqs + 1) * 4);                                       // Cumulative lengths
    
    // Create proper summary
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      return;
    }

    // Total tokens for throughput calculation
    state.add_element_count(total_tokens, "Tokens");

    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto result = mha_varlen_fwd(
        q,
        k,
        v,
        out_,
        cu_seqlens,
        cu_seqlens,
        seqused_k,
        leftpad_k_,
        block_table_,
        alibi_slopes_,
        seq_len,
        seq_len,
        dropout_p,
        softmax_scale,
        zero_tensors,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        return_softmax,
        gen_
      );
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      }
      timer.stop();
    });
    
  } catch (const c10::Error& e) {
    state.skip("OOM: " + std::string(e.what()));
    return;
  }
}

#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)
// Variable Length Benchmark
NVBENCH_BENCH(run_mha_varlen_fwd)
    .set_name("run_mha_varlen_fwd")
    .add_int64_axis("num_seqs", {1, 8, 32, 64})  // Added single sequence option
    .add_int64_axis("seq_len", {128, 512, 1024, 2048, 4096, 8192, 16384})  // Added longer sequences
    .add_int64_axis("num_heads", {12, 16, 32, 40, 128})  // Added 12, 40, 128
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 128})  // Added 10, 12, 128
    .add_int64_axis("head_size", {56, 64, 80, 128});  // Added 56
