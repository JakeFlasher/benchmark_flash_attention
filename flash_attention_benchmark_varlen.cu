#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include <sstream>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"

constexpr int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

// Variable Length Attention Benchmark
void run_mha_varlen_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  float dropout_p = state.get_float64_or_default("dropout", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;
  bool zero_tensors = state.get_int64_or_default("zero_tensors", 0) != 0;

  // Operation description
  if (state.get_int64_or_default("verbose", 0)) {
    auto& op_summ = state.add_summary("operation_details");
    op_summ.set_string("name", "Variable Length Attention");
    
    std::stringstream desc_ss;
    desc_ss << "Packed Q[" << num_seqs * seq_len << "×" << head_size 
            << "] with " << num_seqs << " sequences";
    op_summ.set_string("description", desc_ss.str());
    
    if (is_causal) op_summ.set_string("mask", "causal");
    
    if (window_size_left > 0 || window_size_right > 0) {
      std::stringstream window_ss;
      window_ss << window_size_left << ":" << window_size_right;
      op_summ.set_string("window", window_ss.str());
    }
    
    op_summ.set_float64("gqa_ratio", static_cast<double>(num_heads) / num_kv_heads);
  }

  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat16);
  auto int_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32);
  
  try {
    // For variable length sequences, we pack them into a single tensor
    int total_tokens = num_seqs * seq_len;
    
    // Create packed tensors
    auto q = torch::rand({total_tokens, num_heads, head_size}, tensor_options);
    auto k = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    auto v = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    
    // Create cumulative sequence lengths tensor
    auto cu_seqlens = torch::zeros({num_seqs + 1}, int_options);
    for (int i = 1; i <= num_seqs; i++) {
      cu_seqlens[i] = i * seq_len;
    }
    
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> seqused_k = std::nullopt;
    c10::optional<const at::Tensor> leftpad_k_ = std::nullopt;
    c10::optional<at::Tensor> block_table_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    c10::optional<at::Generator> gen_ = std::nullopt;
    
    // Memory operation and throughput tracking
    int sizeof_half = 2;
    int64_t q_size = total_tokens * num_heads * head_size * sizeof_half;
    int64_t k_size = total_tokens * num_kv_heads * head_size * sizeof_half;
    int64_t v_size = total_tokens * num_kv_heads * head_size * sizeof_half;
    int64_t out_size = total_tokens * num_heads * head_size * sizeof_half;
    int64_t overhead_size = (num_seqs + 1) * 4; // cu_seqlens
    
    // Add memory reads/writes
    state.add_global_memory_reads<char>(q_size, "Q Tensor");
    state.add_global_memory_reads<char>(k_size, "K Tensor");
    state.add_global_memory_reads<char>(v_size, "V Tensor");
    state.add_global_memory_writes<char>(out_size, "Output");
    
    // Token throughput
    state.add_element_count(total_tokens, "Tokens");
    
    // FLOPS estimation
    int64_t flops = 2 * total_tokens * seq_len * num_heads * head_size;
    auto& flops_summ = state.add_summary("flops");
    flops_summ.set_string("name", "Est. FLOPS");
    flops_summ.set_string("description", "Estimated floating point operations");
    flops_summ.set_int64("value", flops);
    
    // Memory usage estimation
    int64_t est_memory_usage = q_size + k_size + v_size + out_size + overhead_size;
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      std::string error_msg = "CUDA error: ";
      error_msg += hipGetErrorString(status);
      state.skip(error_msg);
      return;
    }

    // Execute benchmark with proper timing
    state.exec([&](nvbench::launch& launch) {
      auto result = mha_varlen_fwd(
        q,
        k,
        v,
        out_,
        cu_seqlens,
        cu_seqlens,
        seqused_k,
        leftpad_k_,
        block_table_,
        alibi_slopes_,
        seq_len,
        seq_len,
        dropout_p,
        softmax_scale,
        zero_tensors,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        return_softmax,
        gen_
      );
      
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        std::string error_msg = "CUDA error: ";
        error_msg += hipGetErrorString(status);
        state.skip(error_msg);
      }
    });
    
  } catch (const c10::Error& e) {
    std::string error_msg = "OOM: ";
    error_msg += e.what();
    state.skip(error_msg);
    return;
  }
}

// Register benchmark with expanded configuration parameters
NVBENCH_BENCH(run_mha_varlen_fwd)
    .set_name("run_mha_varlen_fwd")
    .add_int64_axis("num_seqs", {1, 8, 32, 64})
    .add_int64_axis("seq_len", {128, 512, 1024, 2048, 4096, 8192, 16384, 32768})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 128})
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 128})
    .add_int64_axis("head_size", {56, 64, 80, 128})
    .add_int64_axis("window_left", {-1, 128, 1024})
    .add_int64_axis("window_right", {-1, 128, 1024})
    .add_int64_axis("causal", {0, 1});
