#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include "thrust/device_vector.h"
#include "nvbench/nvbench.cuh"
#include "flash_api.h"
// ===== Variable Length Attention Forward Pass Benchmark =====
void run_mha_varlen_fwd(nvbench::state& state) {
  hipStream_t torch_stream = at::cuda::getCurrentCUDAStream();
  state.set_cuda_stream(nvbench::make_cuda_stream_view(torch_stream));

  int64_t num_seqs = state.get_int64("num_seqs");
  int64_t seq_len = state.get_int64("seq_len");
  int64_t num_heads = state.get_int64("num_heads");
  int64_t num_kv_heads = state.get_int64("num_kv_heads");
  int64_t head_size = state.get_int64("head_size");
  
  // Get optional parameters with defaults
  bool is_causal = state.get_int64_or_default("causal", 1) != 0;
  float softmax_scale = state.get_float64_or_default("scale", 1.0f / sqrt(head_size));
  float softcap = state.get_float64_or_default("softcap", 0.0f);
  float dropout_p = state.get_float64_or_default("dropout", 0.0f);
  int64_t window_size_left = state.get_int64_or_default("window_left", -1);
  int64_t window_size_right = state.get_int64_or_default("window_right", -1);
  bool return_softmax = state.get_int64_or_default("return_softmax", 0) != 0;
  bool zero_tensors = state.get_int64_or_default("zero_tensors", 0) != 0;
  
  // Select data type based on parameter
  auto dtype = state.get_string_or_default("dtype", "float16") == "bfloat16" 
               ? torch::kBFloat16 : torch::kFloat16;
  
  auto tensor_options = torch::TensorOptions().device(torch::kCUDA).dtype(dtype);
  auto int_options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kInt32);
  
  try {
    // Variable length sequences setup
    // For this benchmark, we'll create a more realistic variable length scenario
    // where sequences have different lengths
    std::vector<int> sequence_lengths;
    int64_t total_tokens = 0;
    
    if (state.get_int64_or_default("var_lengths", 0) != 0) {
      // Create varying sequence lengths with a distribution based on seq_len parameter
      std::mt19937 rng(42);  // Fixed seed for reproducibility
      std::normal_distribution<> dist(seq_len, seq_len/4);
      
      for (int i = 0; i < num_seqs; i++) {
        int length = std::max(1, std::min(static_cast<int>(seq_len*2), static_cast<int>(dist(rng))));
        sequence_lengths.push_back(length);
        total_tokens += length;
      }
    } else {
      // Fixed length for all sequences
      for (int i = 0; i < num_seqs; i++) {
        sequence_lengths.push_back(seq_len);
      }
      total_tokens = num_seqs * seq_len;
    }
    
    // Create cumulative sequence lengths tensor for Flash Attention API
    std::vector<int> cu_seqlens_vec(num_seqs + 1, 0);
    for (int i = 0; i < num_seqs; i++) {
      cu_seqlens_vec[i+1] = cu_seqlens_vec[i] + sequence_lengths[i];
    }
    
    auto cu_seqlens = torch::from_blob(cu_seqlens_vec.data(), 
                                      {static_cast<int64_t>(cu_seqlens_vec.size())}, 
                                      int_options.clone());
    
    // Create packed tensors with correct total token count
    auto q = torch::rand({total_tokens, num_heads, head_size}, tensor_options);
    auto k = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    auto v = torch::rand({total_tokens, num_kv_heads, head_size}, tensor_options);
    
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> seqused_k = std::nullopt;
    c10::optional<const at::Tensor> leftpad_k_ = std::nullopt;
    c10::optional<at::Tensor> block_table_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    c10::optional<at::Generator> gen_ = std::nullopt;
    
    // Create alibi slopes if testing that attention variant
    if (state.get_int64_or_default("use_alibi", 0) != 0) {
      auto slopes = torch::linspace(1.0f/8, 1.0f, num_heads, 
                                   torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat32));
      alibi_slopes_ = slopes;
    }
    
    // Calculate memory usage for reporting
    int element_size = 2;  // Both FP16 and BF16 are 2 bytes
    int64_t o_write_elements = total_tokens * num_heads * head_size;
    int64_t q_read_elements = total_tokens * num_heads * head_size;
    int64_t k_read_elements = total_tokens * num_kv_heads * head_size;
    int64_t v_read_elements = total_tokens * num_kv_heads * head_size;
    
    // Add memory reads and writes for throughput reporting
    if (dtype == torch::kFloat16) {
      state.add_global_memory_reads<at::Half>(q_read_elements + k_read_elements + v_read_elements);
      state.add_global_memory_writes<at::Half>(o_write_elements);
    } else {
      state.add_global_memory_reads<at::BFloat16>(q_read_elements + k_read_elements + v_read_elements);
      state.add_global_memory_writes<at::BFloat16>(o_write_elements);
    }
    
    // Memory usage estimation for custom summary
    int64_t est_memory_usage = 
        (q_read_elements + k_read_elements + v_read_elements + o_write_elements) * element_size +
        ((num_seqs + 1) * 4);  // cu_seqlens

    if (alibi_slopes_.has_value()) {
      est_memory_usage += alibi_slopes_.value().numel() * sizeof(float);
    }
    
    auto& mem_summary = state.add_summary("memory_usage");
    mem_summary.set_string("name", "Memory Usage");
    mem_summary.set_string("description", "Estimated memory used in MiB");
    mem_summary.set_float64("value", est_memory_usage / (1024.0 * 1024.0));
    
    auto status = hipGetLastError();
    if (status != hipSuccess) {
      state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      return;
    }

    // Report tokens for throughput calculation
    state.add_element_count(total_tokens, "Tokens");

    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto result = mha_varlen_fwd(
        q,
        k,
        v,
        out_,
        cu_seqlens,
        cu_seqlens,
        seqused_k,
        leftpad_k_,
        block_table_,
        alibi_slopes_,
        *std::max_element(sequence_lengths.begin(), sequence_lengths.end()),  // max_seqlen_q
        *std::max_element(sequence_lengths.begin(), sequence_lengths.end()),  // max_seqlen_k
        dropout_p,
        softmax_scale,
        zero_tensors,
        is_causal,
        window_size_left,
        window_size_right,
        softcap,
        return_softmax,
        gen_
      );
      auto status = hipGetLastError();
      if (status != hipSuccess) {
        state.skip("CUDA error: " + std::string(hipGetErrorString(status)));
      }
      timer.stop();
    });
    
  } catch (const c10::Error& e) {
    state.skip("OOM: " + std::string(e.what()));
    return;
  }
}



#define TO_STRING_IND(x) #x
#define TO_STRING(x) TO_STRING_IND(x)
// Variable Length Benchmark
NVBENCH_BENCH(run_mha_varlen_fwd)
    .set_name("run_mha_varlen_fwd")
    .add_int64_axis("num_seqs", {1, 8, 16, 32, 64})
    .add_int64_axis("seq_len", {128, 512, 1024, 2048, 4096, 8192, 16384})
    .add_int64_axis("num_heads", {12, 16, 32, 40, 64, 128})
    .add_int64_axis("num_kv_heads", {4, 8, 10, 12, 16, 32, 128})
    .add_int64_axis("head_size", {56, 64, 80, 96, 128, 160})
    .add_string_axis("dtype", {"float16", "bfloat16"})
    .add_int64_axis("var_lengths", {0, 1})  // Test both fixed and variable lengths
    .add_int64_axis("use_alibi", {0, 1});   // Test with and without ALiBi
